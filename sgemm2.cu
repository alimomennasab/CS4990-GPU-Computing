
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h> 

#define CHECK(call){ \
    const hipError_t cuda_ret = call; \
    if (cuda_ret != hipSuccess){ \
        printf("Error: %s:%d, ", __FILE__, __LINE__); \
        printf("code: %d, reason:%s\n", cuda_ret, hipGetErrorString(cuda_ret)); \
        exit(-1); \
    } \
}

double CPUTimer(){
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec/1.0e6);
}

// CPU only matrix multiplication
void basicSgemm_h(int m, int k, int n, const float *A_h, const float *B_h, float* C_h){
    double startTime = CPUTimer();
    for (int row1 = 0; row1 < m; row1++){
        for (int col2 = 0; col2 < n; col2++){
            C_h[row1 * n + col2] = (float)0;
            for (int col1AndRow2 = 0; col1AndRow2 < k; col1AndRow2++){
                C_h[row1 * n + col2] += A_h[row1 * k + col1AndRow2] * B_h[col1AndRow2 * n + col2];
            }
        }
    }

    double endTime = CPUTimer();
    printf("basicSgemm_h on CPU: %.6f s\n\n", endTime - startTime);
}

// CUDA kernel where each thread computes one output matrix element
__global__ void matrixMulKernel_1thread1element (int m, int k, int n, const float *A_d, const float *B_d, float* C_d){
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n){
        float sum = 0.0f;
        for (unsigned int i = 0; i < k; ++i) {
            sum += A_d[row * k + i] * B_d[i * n + col];
        }
        C_d[row * n + col] = sum;
    }
}

// A CUDA kernel for a “tiled” version of matrix multiplication, which uses dynamically allocated space in shared memory. 
// Here we assume each thread calculates one element of the output matrix
__global__ void matrixMulKernel_tiled(int m, int k, int n, const float *A_d, const float 
    *B_d, float* C_d, unsigned Adz_sz, unsigned Bdz_sz){

}

// host function allocating & freeing & copying memory/calling/timing for matrixMulKernel_1thread1element
void basicSgemm_d_1thread1element (int m, int k, int n, const float *A_h, const float *B_h, float* C_h){
    printf("1thread1element on GPU: \n");
    double startTotalTime = CPUTimer();

    // allocate device memory on GPU for arrays A_d, B_d, C_d
    float *A_d,  *B_d, *C_d;
    double startTimeCudaMalloc = CPUTimer();
    CHECK(hipMalloc((void**)&A_d, sizeof(float)*(m * k)));
    CHECK(hipMalloc((void**)&B_d, sizeof(float)*(k * n)));
    CHECK(hipMalloc((void**)&C_d, sizeof(float)*(m * n)));
    double endTimeCudaMalloc = CPUTimer();
    printf("    1thread1element cudaMalloc: %.6f s\n", endTimeCudaMalloc - startTimeCudaMalloc);

    // copy A_h and B_h to A_d and B_d
    double startTimeCudaMemcpy = CPUTimer();
    CHECK(hipMemcpy(A_d, A_h, sizeof(float)*(m*k), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(B_d, B_h, sizeof(float)*(k*n), hipMemcpyHostToDevice));
    double endTimeCudaMemcpy = CPUTimer();
    printf("    1thread1element cudaMemcpy: %.6f s\n", endTimeCudaMemcpy - startTimeCudaMemcpy);

    // calling matrixMulKernel_1thread1element kernel
    dim3 blockDim(32, 32);
    unsigned int gridDimX = (n + blockDim.x - 1) / blockDim.x;  // number of blocks in x-direction (cols)
    unsigned int gridDimY = (m + blockDim.y - 1) / blockDim.y;  // number of blocks in y-direction (rows)
    dim3 gridDim = {gridDimX, gridDimY};

    double startTimeKernelCall = CPUTimer();
    matrixMulKernel_1thread1element<<<gridDim, blockDim>>>(m, k, n, A_d, B_d, C_d);
    double endTimeKernelCall = CPUTimer();
    printf("    matrixMulKernel_1thread1element<<<(%d,%d,1),(%d,%d,1)>>> call time: %.6f s\n", 
        gridDim.x, gridDim.y, blockDim.x, blockDim.y, endTimeKernelCall - startTimeKernelCall);

    // copy GPU matmul results to host memory
    double startTimeCudaMemcpyResults = CPUTimer();
    hipMemcpy(C_h, C_d, sizeof(float) * m * n, hipMemcpyDeviceToHost);
    double endTimeCudaMemcpyResults = CPUTimer();
    printf("    1thread1element results cudaMemcpy: %.6f s\n", endTimeCudaMemcpyResults - startTimeCudaMemcpyResults);

    // free device memory
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

    // timing results
    double endTotalTime = CPUTimer();
    printf("    1thread1element total time: %.6f s\n", endTotalTime - startTotalTime);
}

// host function allocating & freeing & copying memory/calling/timing formatrixMulKernel_tiled()
void basicSgemm_d_tiled (int m, int k, int n, const float *A_h, const float *B_h, float*C_h){

}

// verification between GPU and CPU result matrices
bool verify(float* CPU_Answer, float* GPU_Answer, unsigned int nRows, unsigned int nCols){
    float epsilon = 0.01f;
    int wrongCount = 0;

    for (unsigned int i = 0; i < nRows; i++) {
        for (unsigned int j = 0; j < nCols; j++) {
            float cpuValue = CPU_Answer[i * nCols + j];
            float gpuValue = GPU_Answer[i * nCols + j];
            float difference = fabs(cpuValue - gpuValue);
            if (difference > epsilon) {
                printf("cpuValue[%d][%d] = %f doesn't match gpuValue[%d][%d] = %f \n", i, j, cpuValue, i, j, gpuValue);
                printf("difference: %f \n", difference);
                wrongCount++;
            }
        }
    }
    if (wrongCount != 0){
        printf("Mismatching CPU/GPU result matrix elements count: %d \n\n", wrongCount);
        return false;
    }
    return true;
}

// main
int main(int argc, char *argv[]) {
    // ./sgemm <m> <k> <n>
    CHECK(hipDeviceSynchronize()); 

    int m = atof(argv[1]);
    int k = atof(argv[2]);
    int n = atof(argv[3]);

    printf("Vector size of matrix A (m * k): %d * %d = %d\n", m, k, m * k);
    printf("Vector size of matrix B (k * n): %d * %d = %d\n", k, n, k * n);
    printf("Vector size of matrix C (m * n): %d * %d = %d\n\n", m, n, m * n);

    // initialize and populate arrays for A_h, B_h, and C_h with host memory
    float* A_h = (float*) malloc(sizeof(float)*(m * k));
    float* B_h = (float*) malloc(sizeof(float)*(k * n));
    float* C_h = (float*) calloc(m * n, sizeof(float));
    float* C_h_gpu_answer = (float*) calloc(m * n, sizeof(float));

    for (unsigned int i = 0; i < m * k; i++) {
        A_h[i] = rand()%100/100.0f;
    }
    for (unsigned int i = 0; i < k * n; i++) {
        B_h[i] = rand()%100/100.0f;
    }

    // perform CPU matrix multiplication
    basicSgemm_h(m, k, n, A_h, B_h, C_h);

    // perform GPU matrix multiplication
    basicSgemm_d_1thread1element(m, k, n, A_h, B_h, C_h_gpu_answer);
    if (verify(C_h, C_h_gpu_answer, m, n)){
        printf("Verification successful: GPU and CPU result matrices match\n\n");
    } else {
        printf("Verification failed: GPU and CPU result matrices don't match\n\n");
    }

    // perform GPU tiled matrix multiplication

    // free allocated host memory
    free(A_h);
    free(B_h);
    free(C_h);
    free(C_h_gpu_answer);

    return 0;
}
